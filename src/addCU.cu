/**
 *
 * @brief Add function GPU implementation
 *
 * @file addCU.cu
 * @author Guillermo Hernández
 * @date 16 Mar 2016
 *
 */

// System includes
#include <stdlib.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>



/**
 * @brief Macro to check for CUDA errors
 *
 * If code!=cudaSuccess (0) it prints a message in stderr and returns 1.
 *
 * @param code integer code returned by last CUDA-related function (cudaMalloc, cudaGetLastError,...)
 * @param msg a string describing the error
 */
#define checkError(code,msg) if (code != hipSuccess) {\
		fprintf(stderr, msg);\
		fprintf(stderr,"(error code %s)\n",hipGetErrorString(err));\
		return 1;\
	}

/**
 * @brief CUDA Kernel to calculate vector addition
 *
 * Kernel to computes the vector addition of @p A and @p B into @p C, all of them having @p n elements
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}



extern "C"
int AddGPU(const float *h_A, const float *h_B,float *h_C, int n)
{
	// GPU implementation must wrap the call to the kernel

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;


	size_t size = n * sizeof(float);
	// Allocate the device input vectors
	float *d_A = NULL;
	err = hipMalloc((void **) &d_A, size);
	checkError(err,"Failed to allocate device vector A");


	float *d_B = NULL;
	err = hipMalloc((void **) &d_B, size);
	checkError(err,"Failed to allocate device vector B");



	float *d_C = NULL;
	err = hipMalloc((void **) &d_C, size);
	checkError(err,"Failed to allocate device vector C");


	// Copy input to device memory
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	checkError(err,"Failed to copy vector A from host to device");

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	checkError(err,"Failed to copy vector B from host to device");

	// Launch the kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
			threadsPerBlock);
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
	err = hipGetLastError();
	checkError(err,"Failed to launch vectorAdd kernel");


	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	checkError(err,"Failed to copy vector C from device to host");


	// Free device global memory
	err = hipFree(d_A);
	checkError(err,"Failed to free device vector A");

	err = hipFree(d_B);
	checkError(err,"Failed to free device vector B");

	err = hipFree(d_C);
	checkError(err,"Failed to free device vector C");



	return 0;
}
